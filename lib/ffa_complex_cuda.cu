#include "hip/hip_runtime.h"
#include "loki/algorithms/ffa_complex.hpp"

#include <memory>
#include <stdexcept>

#include <spdlog/spdlog.h>

#include <hip/hip_runtime_api.h>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/transform.h>

#include "loki/algorithms/fold.hpp"
#include "loki/common/types.hpp"
#include "loki/cuda_utils.cuh"
#include "loki/plans_cuda.cuh"
#include "loki/utils/fft.hpp"

namespace loki::algorithms {

namespace {
// OPTIMIZED: One thread per smallest work unit, optimized for memory coalescing
__global__ void
kernel_ffa_complex_iter(const ComplexTypeCUDA* __restrict__ fold_in,
                        ComplexTypeCUDA* __restrict__ fold_out,
                        const plans::FFACoordDPtrs coords,
                        int ncoords_cur,
                        int ncoords_prev,
                        int nsegments,
                        int nbins_f,
                        int nbins) {

    // 1D thread mapping with optimal work distribution
    const auto tid = static_cast<int>((blockIdx.x * blockDim.x) + threadIdx.x);
    const auto total_work = ncoords_cur * nsegments * nbins_f;

    if (tid >= total_work) {
        return;
    }

    // Decode thread ID to (icoord, iseg, k) - OPTIMIZED ORDER for coalescing
    const int k      = tid % nbins_f; // Frequency bin (fastest varying)
    const int temp   = tid / nbins_f;
    const int iseg   = temp % nsegments;
    const int icoord = temp / nsegments;

    // Precompute coordinate data (avoid repeated access)
    const auto coord_tail = coords.i_tail[icoord];
    const auto coord_head = coords.i_head[icoord];
    const auto shift_tail = coords.shift_tail[icoord];
    const auto shift_head = coords.shift_head[icoord];

    // Precompute phase factors: exp(-2πi * k * shift / nbins)
    const auto phase_factor_tail =
        static_cast<float>(-2.0F * M_PI * k * shift_tail / nbins);
    const auto phase_factor_head =
        static_cast<float>(-2.0F * M_PI * k * shift_head / nbins);
    // Fast sincos computation
    float cos_tail, sin_tail, cos_head, sin_head;
    __sincosf(phase_factor_tail, &sin_tail, &cos_tail);
    __sincosf(phase_factor_head, &sin_head, &cos_head);

    // Calculate memory offsets for e and v components
    const int tail_offset_e =
        ((iseg * 2) * ncoords_prev * 2 * nbins_f) + (coord_tail * 2 * nbins_f);
    const int tail_offset_v = tail_offset_e + nbins_f;
    const int head_offset_e = ((iseg * 2 + 1) * ncoords_prev * 2 * nbins_f) +
                              (coord_head * 2 * nbins_f);
    const int head_offset_v = head_offset_e + nbins_f;

    const int out_offset_e =
        (iseg * ncoords_cur * 2 * nbins_f) + (icoord * 2 * nbins_f);
    const int out_offset_v = out_offset_e + nbins_f;

    // Load complex values for both e and v components
    const ComplexTypeCUDA data_tail_e = fold_in[tail_offset_e + k];
    const ComplexTypeCUDA data_tail_v = fold_in[tail_offset_v + k];
    const ComplexTypeCUDA data_head_e = fold_in[head_offset_e + k];
    const ComplexTypeCUDA data_head_v = fold_in[head_offset_v + k];

    // OPTIMIZED complex multiplication using fmaf
    // tail_shifted_e = data_tail_e * exp(-2πi * k * shift_tail / nbins)
    const float real_tail_e =
        fmaf(data_tail_e.real(), cos_tail, -data_tail_e.imag() * sin_tail);
    const float imag_tail_e =
        fmaf(data_tail_e.real(), sin_tail, data_tail_e.imag() * cos_tail);
    const float real_head_e =
        fmaf(data_head_e.real(), cos_head, -data_head_e.imag() * sin_head);
    const float imag_head_e =
        fmaf(data_head_e.real(), sin_head, data_head_e.imag() * cos_head);
    const float real_tail_v =
        fmaf(data_tail_v.real(), cos_tail, -data_tail_v.imag() * sin_tail);
    const float imag_tail_v =
        fmaf(data_tail_v.real(), sin_tail, data_tail_v.imag() * cos_tail);
    const float real_head_v =
        fmaf(data_head_v.real(), cos_head, -data_head_v.imag() * sin_head);
    const float imag_head_v =
        fmaf(data_head_v.real(), sin_head, data_head_v.imag() * cos_head);
    // Complex addition and store results
    fold_out[out_offset_e + k] =
        ComplexTypeCUDA(real_tail_e + real_head_e, imag_tail_e + imag_head_e);

    fold_out[out_offset_v + k] =
        ComplexTypeCUDA(real_tail_v + real_head_v, imag_tail_v + imag_head_v);
}
} // namespace

class FFACOMPLEXCUDA::Impl {
public:
    Impl(search::PulsarSearchConfig cfg, int device_id)
        : m_cfg(std::move(cfg)),
          m_ffa_plan(m_cfg),
          m_device_id(device_id) {
        cuda_utils::set_device(m_device_id);
        // Allocate memory for the FFA buffers
        m_fold_in_d.resize(m_ffa_plan.get_buffer_size_complex(),
                           ComplexTypeCUDA(0.0F, 0.0F));
        m_fold_out_d.resize(m_ffa_plan.get_buffer_size_complex(),
                            ComplexTypeCUDA(0.0F, 0.0F));

        // Initialize the brute fold
        const auto t_ref =
            m_cfg.get_nparams() == 1 ? 0.0 : m_ffa_plan.tsegments[0] / 2.0;
        const auto freqs_arr = m_ffa_plan.params[0].back();

        m_the_bf = std::make_unique<algorithms::BruteFoldCUDA>(
            freqs_arr, m_ffa_plan.segment_lens[0], m_cfg.get_nbins(),
            m_cfg.get_nsamps(), m_cfg.get_tsamp(), t_ref, m_device_id);
        m_fold_in_tmp_d.resize(m_the_bf->get_fold_size(), 0.0F);

        plans::transfer_ffa_plan_to_device(m_ffa_plan, m_ffa_plan_d);
        cuda_utils::check_last_cuda_error(
            "FFACUDA::Impl::transfer_ffa_plan_to_device failed");
    }

    ~Impl()                      = default;
    Impl(const Impl&)            = delete;
    Impl& operator=(const Impl&) = delete;
    Impl(Impl&&)                 = delete;
    Impl& operator=(Impl&&)      = delete;

    const plans::FFAPlan& get_plan() const { return m_ffa_plan; }

    void execute_h(std::span<const float> ts_e,
                   std::span<const float> ts_v,
                   std::span<float> fold) {
        check_inputs(ts_e.size(), ts_v.size(), fold.size());

        // Resize buffers only if needed
        if (m_ts_e_d.size() != ts_e.size()) {
            m_ts_e_d.resize(ts_e.size());
            m_ts_v_d.resize(ts_v.size());
        }
        if (m_fold_output_d.size() != fold.size()) {
            m_fold_output_d.resize(fold.size());
        }
        // Copy input data to device
        hipStream_t stream = nullptr;
        hipMemcpyAsync(thrust::raw_pointer_cast(m_ts_e_d.data()), ts_e.data(),
                        ts_e.size() * sizeof(float), hipMemcpyHostToDevice,
                        stream);
        hipMemcpyAsync(thrust::raw_pointer_cast(m_ts_v_d.data()), ts_v.data(),
                        ts_v.size() * sizeof(float), hipMemcpyHostToDevice,
                        stream);

        // Execute FFA on device using persistent buffers
        execute_d(
            cuda::std::span<const float>(
                thrust::raw_pointer_cast(m_ts_e_d.data()), m_ts_e_d.size()),
            cuda::std::span<const float>(
                thrust::raw_pointer_cast(m_ts_v_d.data()), m_ts_v_d.size()),
            cuda::std::span<float>(
                thrust::raw_pointer_cast(m_fold_output_d.data()),
                m_fold_output_d.size()),
            stream);

        // Copy result back to host
        thrust::copy(m_fold_output_d.begin(), m_fold_output_d.end(),
                     fold.begin());
        spdlog::debug("FFACUDA::Impl: Host execution complete");
    }

    void execute_d(cuda::std::span<const float> ts_e_d,
                   cuda::std::span<const float> ts_v_d,
                   cuda::std::span<float> fold_d,
                   hipStream_t stream) {
        check_inputs(ts_e_d.size(), ts_v_d.size(), fold_d.size());
        thrust::device_vector<ComplexTypeCUDA> fold_d_complex(
            m_ffa_plan.get_fold_size_complex(), ComplexTypeCUDA(0.0F, 0.0F));
        cuda::std::span<ComplexTypeCUDA> fold_d_complex_span(
            thrust::raw_pointer_cast(fold_d_complex.data()),
            fold_d_complex.size());
        execute_device(ts_e_d, ts_v_d, fold_d_complex_span, stream);
        // IRFFT the output
        const auto nfft = m_ffa_plan.get_fold_size() / m_cfg.get_nbins();
        utils::irfft_batch_cuda(fold_d_complex_span, fold_d,
                                static_cast<int>(nfft),
                                static_cast<int>(m_cfg.get_nbins()), stream);
        spdlog::debug("FFACUDA::Impl: Device execution complete on stream");
    }

    void execute_d(cuda::std::span<const float> ts_e_d,
                   cuda::std::span<const float> ts_v_d,
                   cuda::std::span<ComplexTypeCUDA> fold_d,
                   hipStream_t stream) {
        check_inputs_complex(ts_e_d.size(), ts_v_d.size(), fold_d.size());
        execute_device(ts_e_d, ts_v_d, fold_d, stream);
        spdlog::debug("FFACUDA::Impl: Device execution complete on stream");
    }

private:
    search::PulsarSearchConfig m_cfg;
    plans::FFAPlan m_ffa_plan;
    plans::FFAPlanD m_ffa_plan_d;
    int m_device_id;

    // Buffers for the FFA plan
    thrust::device_vector<ComplexTypeCUDA> m_fold_in_d;
    thrust::device_vector<ComplexTypeCUDA> m_fold_out_d;
    thrust::device_vector<float> m_fold_in_tmp_d;

    // Add persistent input/output buffers
    thrust::device_vector<float> m_ts_e_d;
    thrust::device_vector<float> m_ts_v_d;
    thrust::device_vector<float> m_fold_output_d;

    std::unique_ptr<algorithms::BruteFoldCUDA> m_the_bf;

    void check_inputs(loki::SizeType ts_e_size,
                      loki::SizeType ts_v_size,
                      loki::SizeType fold_size) const {
        if (ts_e_size != m_cfg.get_nsamps()) {
            throw std::runtime_error(
                std::format("FFACOMPLEXCUDA::Impl: ts must have size nsamps. "
                            "Expected {}, got {}",
                            m_cfg.get_nsamps(), ts_e_size));
        }
        if (ts_v_size != ts_e_size) {
            throw std::runtime_error(std::format(
                "FFACOMPLEXCUDA::Impl: ts variance must have size nsamps. "
                "Expected {}, got {}",
                ts_e_size, ts_v_size));
        }
        if (fold_size != m_ffa_plan.get_fold_size()) {
            throw std::runtime_error(std::format(
                "FFACOMPLEXCUDA::Impl: Output array has wrong size. "
                "Expected {}, got {}",
                m_ffa_plan.get_fold_size(), fold_size));
        }
    }

    void check_inputs_complex(loki::SizeType ts_e_size,
                              loki::SizeType ts_v_size,
                              loki::SizeType fold_complex_size) const {
        if (ts_e_size != m_cfg.get_nsamps()) {
            throw std::runtime_error(
                std::format("FFACOMPLEXCUDA::Impl: ts_e must have size nsamps. "
                            "Expected {}, got {}",
                            m_cfg.get_nsamps(), ts_e_size));
        }
        if (ts_v_size != ts_e_size) {
            throw std::runtime_error(
                std::format("FFACOMPLEXCUDA::Impl: ts_v must have size nsamps. "
                            "Expected {}, got {}",
                            ts_e_size, ts_v_size));
        }
        if (fold_complex_size != m_ffa_plan.get_fold_size_complex()) {
            throw std::runtime_error(std::format(
                "FFACOMPLEXCUDA::Impl: Output array has wrong size. "
                "Expected {}, got {}",
                m_ffa_plan.get_fold_size_complex(), fold_complex_size));
        }
    }

    void initialize_device(cuda::std::span<const float> ts_e_d,
                           cuda::std::span<const float> ts_v_d,
                           hipStream_t stream) {
        m_the_bf->execute(
            ts_e_d, ts_v_d,
            cuda::std::span(thrust::raw_pointer_cast(m_fold_in_tmp_d.data()),
                            m_fold_in_tmp_d.size()),
            stream);

        // RFFT the input
        const auto nfft         = m_the_bf->get_fold_size() / m_cfg.get_nbins();
        const auto complex_size = nfft * ((m_cfg.get_nbins() / 2) + 1);
        utils::rfft_batch_cuda(
            cuda::std::span<float>(
                thrust::raw_pointer_cast(m_fold_in_tmp_d.data()),
                m_fold_in_tmp_d.size()),
            cuda::std::span<ComplexTypeCUDA>(
                thrust::raw_pointer_cast(m_fold_in_d.data()), complex_size),
            static_cast<int>(nfft), static_cast<int>(m_cfg.get_nbins()),
            stream);
    }

    void execute_device(cuda::std::span<const float> ts_e_d,
                        cuda::std::span<const float> ts_v_d,
                        cuda::std::span<ComplexTypeCUDA> fold_d_complex,
                        hipStream_t stream) {
        // Clear internal buffers before each execution
        thrust::fill(m_fold_in_d.begin(), m_fold_in_d.end(),
                     ComplexTypeCUDA(0.0F, 0.0F));
        initialize_device(ts_e_d, ts_v_d, stream);

        // Ping-pong between buffers for iterative FFA levels
        ComplexTypeCUDA* fold_in_ptr =
            thrust::raw_pointer_cast(m_fold_in_d.data());
        ComplexTypeCUDA* fold_out_ptr =
            thrust::raw_pointer_cast(m_fold_out_d.data());
        ComplexTypeCUDA* fold_complex_ptr =
            thrust::raw_pointer_cast(fold_d_complex.data());

        const auto levels = m_cfg.get_niters_ffa() + 1;
        auto coords_cur   = m_ffa_plan_d.coordinates.get_raw_ptrs();
        cuda_utils::check_last_cuda_error("thrust::raw_pointer_cast failed");
        coords_cur.update_offsets(m_ffa_plan_d.ncoords[0]);

        // FFA iterations (levels 1 to levels)
        for (SizeType i_level = 1; i_level < levels; ++i_level) {
            const auto nsegments    = m_ffa_plan_d.nsegments[i_level];
            const auto nbins        = m_ffa_plan_d.nbins[i_level];
            const auto ncoords_cur  = m_ffa_plan_d.ncoords[i_level];
            const auto ncoords_prev = m_ffa_plan_d.ncoords[i_level - 1];
            const auto nbins_f      = (nbins / 2) + 1;

            const int total_work = ncoords_cur * nsegments * nbins_f;
            const int block_size = (total_work < 65536) ? 256 : 512;
            const int grid_size  = (total_work + block_size - 1) / block_size;

            const dim3 block_dim(block_size);
            const dim3 grid_dim(grid_size);

            cuda_utils::check_kernel_launch_params(grid_dim, block_dim);

            // Determine output buffer: final iteration writes to fold_d
            ComplexTypeCUDA* current_out_ptr =
                (i_level == levels - 1) ? fold_complex_ptr : fold_out_ptr;

            kernel_ffa_complex_iter<<<grid_dim, block_dim, 0, stream>>>(
                fold_in_ptr, current_out_ptr, coords_cur, ncoords_cur,
                ncoords_prev, nsegments, nbins_f, nbins);
            cuda_utils::check_last_cuda_error(
                "kernel_ffa_complex_iter launch failed");

            // Ping-pong buffers (unless it's the final iteration)
            if (i_level < levels - 1) {
                coords_cur.update_offsets(ncoords_cur);
                std::swap(fold_in_ptr, fold_out_ptr);
            }
        }

        spdlog::debug("FFACUDA::Impl: Iterations submitted to stream.");
    }

}; // End FFACUDA::Impl definition

FFACOMPLEXCUDA::FFACOMPLEXCUDA(const search::PulsarSearchConfig& cfg,
                               int device_id)
    : m_impl(std::make_unique<Impl>(cfg, device_id)) {}

FFACOMPLEXCUDA::~FFACOMPLEXCUDA()                               = default;
FFACOMPLEXCUDA::FFACOMPLEXCUDA(FFACOMPLEXCUDA&& other) noexcept = default;
FFACOMPLEXCUDA&
FFACOMPLEXCUDA::operator=(FFACOMPLEXCUDA&& other) noexcept = default;

const plans::FFAPlan& FFACOMPLEXCUDA::get_plan() const noexcept {
    return m_impl->get_plan();
}

void FFACOMPLEXCUDA::execute(std::span<const float> ts_e,
                             std::span<const float> ts_v,
                             std::span<float> fold) {
    m_impl->execute_h(ts_e, ts_v, fold);
}

void FFACOMPLEXCUDA::execute(cuda::std::span<const float> ts_e,
                             cuda::std::span<const float> ts_v,
                             cuda::std::span<float> fold,
                             hipStream_t stream) {
    m_impl->execute_d(ts_e, ts_v, fold, stream);
}

void FFACOMPLEXCUDA::execute(cuda::std::span<const float> ts_e,
                             cuda::std::span<const float> ts_v,
                             cuda::std::span<ComplexTypeCUDA> fold,
                             hipStream_t stream) {
    m_impl->execute_d(ts_e, ts_v, fold, stream);
}

std::vector<float>
compute_ffa_complex_cuda(std::span<const float> ts_e,
                         std::span<const float> ts_v,
                         const search::PulsarSearchConfig& cfg,
                         int device_id) {
    FFACOMPLEXCUDA ffa(cfg, device_id);
    const auto& ffa_plan = ffa.get_plan();
    std::vector<float> fold(ffa_plan.get_fold_size(), 0.0F);
    ffa.execute(ts_e, ts_v, std::span<float>(fold));
    return fold;
}

} // namespace loki::algorithms