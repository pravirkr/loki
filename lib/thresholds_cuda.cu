#include "hip/hip_runtime.h"
#include "loki/detection/thresholds.hpp"

#include <algorithm>
#include <cassert>
#include <cstdint>
#include <filesystem>
#include <format>
#include <memory>
#include <random>

#include <cuda/std/optional>
#include <cuda/std/span>
#include <hip/hip_runtime.h>
#include <curanddx.hpp>
#include <highfive/highfive.hpp>
#include <spdlog/spdlog.h>
#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>
#include <thrust/host_vector.h>
#include <thrust/partition.h>
#include <thrust/remove.h>
#include <thrust/sequence.h>
#include <thrust/sort.h>
#include <thrust/transform.h>

#include "loki/common/types.hpp"
#include "loki/cuda_utils.cuh"
#include "loki/detection/scheme.hpp"
#include "loki/detection/score.hpp"
#include "loki/math_cuda.cuh"
#include "loki/simulation/simulation.hpp"
#include "loki/utils.hpp"

namespace loki::detection {

// Define the cuRANDDx Generator Descriptor.
using RNG = decltype(curanddx::Generator<curanddx::philox4_32>() +
                     curanddx::PhiloxRounds<10>() +
                     curanddx::SM<CURANDDX_SM>() + curanddx::Thread());

namespace {

/**
 * Device-side handle for CUDA memory - lightweight POD type
 */
struct FoldVectorHandleDevice {
    float* data = nullptr; // Pointer to the fold data in device memory
    SizeType ntrials{};
    SizeType capacity_ntrials{};
    SizeType nbins{};
    float variance{};
    SizeType pool_id{};  // 0 for pool A, 1 for pool B
    SizeType slot_idx{}; // Slot index within the pool

    __device__ __host__ FoldVectorHandleDevice();
    __device__ __host__ SizeType size() const { return ntrials * nbins; }
    __device__ __host__ bool is_valid() const { return data != nullptr; }
};

/**
 * Device-side allocator interface - can be called from kernels
 */
struct DevicePoolAllocator {
    float* pool_a_data;
    float* pool_b_data;
    cuda::std::atomic<int>* free_slots_a;
    cuda::std::atomic<int>* free_slots_b;
    int* next_free_a;
    int* next_free_b;
    SizeType slot_size;
    SizeType max_ntrials;
    SizeType nbins;
    SizeType slots_per_pool;
    int current_out_pool; // 0 for A, 1 for B

    /**
     * Allocate from the current "out" pool - callable from device
     */
    __device__ FoldVectorHandleDevice allocate(SizeType ntrials,
                                               float variance) const {
        FoldVectorHandleDevice handle;

        // Select the appropriate pool
        float* pool_data;
        cuda::std::atomic<int>* free_slots;
        int* next_free;
        int pool_id;

        if (current_out_pool == 0) {
            pool_data  = pool_a_data;
            free_slots = free_slots_a;
            next_free  = next_free_a;
            pool_id    = 0;
        } else {
            pool_data  = pool_b_data;
            free_slots = free_slots_b;
            next_free  = next_free_b;
            pool_id    = 1;
        }

        // Atomically get a free slot
        int slot_idx = atomicAdd(next_free, 1);

        if (slot_idx < static_cast<int>(slots_per_pool)) {
            // Find the actual free slot
            for (int i = 0; i < static_cast<int>(slots_per_pool); ++i) {
                int expected = 1;
                if (free_slots[i].compare_exchange_strong(
                        expected, 0, cuda::std::memory_order_acquire)) {
                    // Successfully claimed this slot
                    handle.data             = pool_data + (i * slot_size);
                    handle.ntrials          = ntrials;
                    handle.capacity_ntrials = max_ntrials;
                    handle.nbins            = nbins;
                    handle.variance         = variance;
                    handle.pool_id          = pool_id;
                    handle.slot_idx         = i;
                    break;
                }
            }
        }

        return handle;
    }

    /**
     * Deallocate - callable from device
     */
    __device__ void deallocate(const FoldVectorHandleDevice& handle) {
        if (!handle.is_valid()) {
            return;
        }

        cuda::std::atomic<int>* free_slots =
            (handle.pool_id == 0) ? free_slots_a : free_slots_b;

        // Mark slot as free
        free_slots[handle.slot_idx].store(1, cuda::std::memory_order_release);
    }
};

/**
 * CUDA Dual-Pool Memory Manager using thrust::device_vector for safety
 */
class DualPoolFoldManagerDevice {
public:
    DualPoolFoldManagerDevice(SizeType nbins,
                              SizeType ntrials_min,
                              SizeType slots_per_pool)
        : m_nbins(nbins),
          m_max_ntrials(2 * ntrials_min),
          m_slot_size(m_max_ntrials * nbins),
          m_slots_per_pool(slots_per_pool) {

        // Allocate device memory pools
        m_pool_a.resize(m_slots_per_pool * m_slot_size);
        m_pool_b.resize(m_slots_per_pool * m_slot_size);

        // Initialize free slot tracking on device
        m_free_slots_a.resize(m_slots_per_pool);
        m_free_slots_b.resize(m_slots_per_pool);

        // Initialize all slots as free (1 = free, 0 = occupied)
        thrust::fill(m_free_slots_a.begin(), m_free_slots_a.end(), 1);
        thrust::fill(m_free_slots_b.begin(), m_free_slots_b.end(), 1);

        // Allocate counters
        hipMalloc(&m_next_free_a, sizeof(int));
        hipMalloc(&m_next_free_b, sizeof(int));
        hipMemset(m_next_free_a, 0, sizeof(int));
        hipMemset(m_next_free_b, 0, sizeof(int));
    }

    ~DualPoolFoldManagerDevice() {
        hipFree(m_next_free_a);
        hipFree(m_next_free_b);
    }

    // Delete copy/move operations
    DualPoolFoldManagerDevice(const DualPoolFoldManagerDevice&) = delete;
    DualPoolFoldManagerDevice&
    operator=(const DualPoolFoldManagerDevice&)                       = delete;
    DualPoolFoldManagerDevice(DualPoolFoldManagerDevice&&)            = delete;
    DualPoolFoldManagerDevice& operator=(DualPoolFoldManagerDevice&&) = delete;

    /**
     * Get device allocator for use in kernels
     */
    DevicePoolAllocator get_device_allocator() {
        return {.pool_a_data  = thrust::raw_pointer_cast(m_pool_a.data()),
                .pool_b_data  = thrust::raw_pointer_cast(m_pool_b.data()),
                .free_slots_a = thrust::raw_pointer_cast(m_free_slots_a.data()),
                .free_slots_b = thrust::raw_pointer_cast(m_free_slots_b.data()),
                .next_free_a  = m_next_free_a,
                .next_free_b  = m_next_free_b,
                .slot_size    = m_slot_size,
                .max_ntrials  = m_max_ntrials,
                .nbins        = m_nbins,
                .slots_per_pool   = m_slots_per_pool,
                .current_out_pool = m_current_out_pool};
    }

    /**
     * Swap pools - must be called from host
     */
    void swap_pools() {
        m_current_out_pool = 1 - m_current_out_pool;

        // Reset the "in" pool for next use
        if (m_current_out_pool == 0) {
            // B is now "in", reset it
            thrust::fill(m_free_slots_b.begin(), m_free_slots_b.end(), 1);
            hipMemset(m_next_free_b, 0, sizeof(int));
        } else {
            // A is now "in", reset it
            thrust::fill(m_free_slots_a.begin(), m_free_slots_a.end(), 1);
            hipMemset(m_next_free_a, 0, sizeof(int));
        }
    }

private:
    thrust::device_vector<float> m_pool_a;
    thrust::device_vector<float> m_pool_b;
    thrust::device_vector<cuda::std::atomic<int>> m_free_slots_a;
    thrust::device_vector<cuda::std::atomic<int>> m_free_slots_b;
    int* m_next_free_a = nullptr;
    int* m_next_free_b = nullptr;

    SizeType m_nbins;
    SizeType m_max_ntrials;
    SizeType m_slot_size;
    SizeType m_slots_per_pool;
    int m_current_out_pool{};
};

struct FoldsTypeDevice {
    FoldVectorHandleDevice folds_h0;
    FoldVectorHandleDevice folds_h1;

    __device__ __host__ FoldsTypeDevice() = default;

    __device__ __host__ FoldsTypeDevice(FoldVectorHandleDevice h0,
                                        FoldVectorHandleDevice h1)
        : folds_h0(h0),
          folds_h1(h1) {}

    __device__ __host__ bool is_empty() const {
        return !folds_h0.is_valid() || !folds_h1.is_valid() ||
               folds_h0.size() == 0 || folds_h1.size() == 0;
    }
};

// Kernel to properly deallocate folds before clearing
struct DeallocateFunctor {
    DevicePoolAllocator allocator;

    __device__ void operator()(cuda::std::optional<FoldsTypeDevice>& fold) {
        if (fold.has_value()) {
            const auto& f = fold.value();
            if (f.folds_h0.is_valid()) {
                allocator.deallocate(f.folds_h0);
            }
            if (f.folds_h1.is_valid()) {
                allocator.deallocate(f.folds_h1);
            }
            fold = cuda::std::nullopt;
        }
    }
};

// Batch transition data for parallel processing
struct TransitionWorkItem { // NOLINT
    int threshold_idx;
    int prob_idx;
    int input_fold_idx;
    StateD input_state;
    float threshold;
    float nbranches;
    FoldsTypeDevice folds_in;
    FoldsTypeDevice folds_sim;
    FoldsTypeDevice folds_pruned;
};

struct TransitionResult { // NOLINT
    int threshold_idx;
    int prob_idx;
    StateD computed_state;
    FoldsTypeDevice folds_out;
    bool invalid;
};

struct TransitionBatch {
    thrust::device_vector<TransitionWorkItem> work_items_d;
    thrust::device_vector<TransitionResult> results_d;

    void reserve(SizeType max_items) {
        work_items_d.reserve(max_items);
        results_d.resize(max_items);
    }
};

__device__ int find_bin_index_device(const float* __restrict__ probs,
                                     int nprobs,
                                     float value) {
    // value below first bin
    if (value < probs[0]) {
        return -1;
    }
    // scan for the first bin > value
    for (int i = 1; i < nprobs; ++i) {
        if (value < probs[i]) {
            return i - 1;
        }
    }
    // value >= last edge
    return nprobs - 1;
}

__device__ void simulate_transition_phase(const TransitionWorkItem& work_item,
                                          const float* __restrict__ profile,
                                          int nbins,
                                          float bias_snr,
                                          float var_add,
                                          uint64_t seed,
                                          uint64_t offset) {
    extern __shared__ float shared_profile_scaled[]; // NOLINT
    const int tid        = static_cast<int>(threadIdx.x);
    const int block_size = static_cast<int>(blockDim.x);
    const int block_id   = static_cast<int>(blockIdx.x);

    // Pre-scale profile for H1 (shared across all threads in block)
    for (int i = tid; i < nbins; i += block_size) {
        shared_profile_scaled[i] = profile[i] * bias_snr;
    }
    __syncthreads();

    // Calculate total elements to process for both H0 and H1
    const int total_elements_h0 =
        static_cast<int>(work_item.folds_sim.folds_h0.ntrials) * nbins;
    const int total_elements_h1 =
        static_cast<int>(work_item.folds_sim.folds_h1.ntrials) * nbins;
    const float noise_stddev = sqrtf(var_add);

    // Lambda for processing 4 elements to match cuRANDDx generate4()
    auto process_batch = [&](int base_i, int total_elements,
                             const float* in_data, float* out_data,
                             int input_ntrials, int seq_id_base,
                             bool add_signal) {
        const int seq_id =
            seq_id_base + ((block_id * total_elements + base_i) / 4);
        // Generate noise using cuRANDDx (Use unique sequence ID for each
        // element)
        RNG rng(seed, seq_id, offset);
        curanddx::normal<float, curanddx::box_muller> dist(0.0F, noise_stddev);
        const float4 noise = dist.generate4(rng);

        int trial_idx   = base_i / nbins;
        int bin_idx     = base_i % nbins;
        int orig_trial  = trial_idx % input_ntrials;
        int orig_offset = (orig_trial * nbins) + bin_idx;

#pragma unroll
        for (int j = 0; j < 4; ++j) {
            const int idx = base_i + j;
            if (idx >= total_elements) {
                break;
            }
            float noise_val = j == 0   ? noise.x
                              : j == 1 ? noise.y  // NOLINT
                              : j == 2 ? noise.z  // NOLINT
                                       : noise.w; // NOLINT
            const float profile_val =
                add_signal ? shared_profile_scaled[bin_idx] : 0.0F;
            out_data[idx] = in_data[orig_offset] + noise_val + profile_val;

            bin_idx++;
            if (bin_idx >= nbins) {
                bin_idx = 0;
                trial_idx++;
                orig_trial = trial_idx % input_ntrials;
            }
            orig_offset = orig_trial * nbins + bin_idx;
        }
    };

    // H0: no signal bias
    for (int base = tid * 4; base < total_elements_h0; base += block_size * 4) {
        process_batch(base, total_elements_h0, work_item.folds_in.folds_h0.data,
                      work_item.folds_sim.folds_h0.data,
                      static_cast<int>(work_item.folds_in.folds_h0.ntrials), 0,
                      false);
    }
    // H1: with signal bias
    const int h1_seq_offset = (total_elements_h0 + 3) / 4;
    for (int base = tid * 4; base < total_elements_h1; base += block_size * 4) {
        process_batch(base, total_elements_h1, work_item.folds_in.folds_h1.data,
                      work_item.folds_sim.folds_h1.data,
                      static_cast<int>(work_item.folds_in.folds_h1.ntrials),
                      h1_seq_offset, true);
    }
}

__device__ float
compute_trial_snr_on_demand(const float* __restrict__ trial_data,
                            int nbins,
                            const SizeType* __restrict__ widths,
                            int nwidths,
                            float stdnoise = 1.0F) {
    const int tid        = static_cast<int>(threadIdx.x);
    const int block_size = static_cast<int>(blockDim.x);
    // Step 1: Compute total sum collaboratively
    float total_sum = 0.0F;
    for (int i = tid; i < nbins; i += block_size) {
        total_sum += trial_data[i];
    }

    // Block-wide reduction for total sum
    for (int offset = block_size / 2; offset > 0; offset >>= 1) {
        total_sum += __shfl_down_sync(0xFFFFFFFF, total_sum, offset);
    }
    // Broadcast to all threads
    total_sum = __shfl_sync(0xFFFFFFFF, total_sum, 0);

    float max_snr = -HIP_INF_F;

    // Step 2: Process each width
    for (int iw = 0; iw < nwidths; ++iw) {
        const int w   = static_cast<int>(widths[iw]);
        const float h = sqrtf(static_cast<float>(nbins - w) /
                              static_cast<float>(nbins * w));
        const float b =
            static_cast<float>(w) * h / static_cast<float>(nbins - w);

        float thread_max_diff = -HIP_INF_F;

        // Each thread processes multiple starting positions
        for (int start = tid; start < nbins; start += block_size) {
            // Compute windowed sum on-the-fly
            float window_sum = 0.0F;

            for (int i = 0; i < w; ++i) {
                int idx = (start + i) % nbins; // Handle circular wrapping
                window_sum += trial_data[idx];
            }

            thread_max_diff = fmaxf(thread_max_diff, window_sum);
        }

        // Block-wide reduction to find maximum difference for this width
        for (int offset = block_size / 2; offset > 0; offset >>= 1) {
            float temp = __shfl_down_sync(0xFFFFFFFF, thread_max_diff, offset);
            thread_max_diff = fmaxf(thread_max_diff, temp);
        }

        if (tid == 0) {
            float snr =
                (((h + b) * thread_max_diff) - (b * total_sum)) / stdnoise;
            max_snr = fmaxf(max_snr, snr);
        }
        __syncthreads();
    }

    // Broadcast final result to all threads
    float result_snr = __shfl_sync(0xFFFFFFFF, max_snr, 0);
    return result_snr;
}

__device__ void
score_and_prune_fused(const TransitionWorkItem& work_item,
                      const SizeType* __restrict__ box_score_widths,
                      int nwidths,
                      int nbins,
                      float threshold,
                      int* output_ntrials_h0,
                      int* output_ntrials_h1) {
    extern __shared__ int shm[]; // NOLINT
    const int tid        = static_cast<int>(threadIdx.x);
    const int block_size = static_cast<int>(blockDim.x);

    // Shared memory for reduction
    int& shared_count_h0   = shm[0];
    int& shared_count_h1   = shm[1];
    int* shared_indices_h0 = &shm[2];
    int* shared_indices_h1 = &shm[2 + block_size];

    if (tid == 0) {
        shared_count_h0 = 0;
        shared_count_h1 = 0;
    }
    __syncthreads();

    // Collect surviving H0 and H1 trials into shared lists
    auto collect = [&](const float* trial_data, int sim_trials, float variance,
                       int& shared_count, int* shared_idx) {
        // Local buffer for good indices
        int local_good[32]; // NOLINT
        int local_count = 0;

        const float stdnoise = sqrtf(variance);

        // strided over trials
        for (int i = tid; i < sim_trials; i += block_size) {
            float score = compute_trial_snr_on_demand(
                trial_data + static_cast<IndexType>(i * nbins), nbins,
                box_score_widths, nwidths, stdnoise);
            if (score > threshold) {
                local_good[local_count++] = i;
                if (local_count == 32) {
                    // Flush local buffer to shared memory
                    int pos = atomicAdd(&shared_count, 32);
#pragma unroll
                    for (int j = 0; j < 32; ++j) {
                        if (pos + j < block_size) {
                            shared_idx[pos + j] = local_good[j];
                        }
                    }
                    local_count = 0;
                }
            }
        }
        // flush remainder
        if (local_count > 0) {
            int pos = atomicAdd(&shared_count, local_count);
#pragma unroll
            for (int j = 0; j < 32; ++j) {
                if (j < local_count && pos + j < block_size) {
                    shared_idx[pos + j] = local_good[j];
                }
            }
        }
    };

    collect(work_item.folds_sim.folds_h0.data,
            static_cast<int>(work_item.folds_sim.folds_h0.ntrials),
            work_item.folds_sim.folds_h0.variance, shared_count_h0,
            shared_indices_h0);
    collect(work_item.folds_sim.folds_h1.data,
            static_cast<int>(work_item.folds_sim.folds_h1.ntrials),
            work_item.folds_sim.folds_h1.variance, shared_count_h1,
            shared_indices_h1);
    __syncthreads();

    // Copy H0 surviving trials
    float* __restrict__ folds_h0_sim    = work_item.folds_sim.folds_h0.data;
    float* __restrict__ folds_h1_sim    = work_item.folds_sim.folds_h1.data;
    float* __restrict__ folds_h0_pruned = work_item.folds_pruned.folds_h0.data;
    float* __restrict__ folds_h1_pruned = work_item.folds_pruned.folds_h1.data;
    for (int i = tid; i < shared_count_h0; i += block_size) {
        const int orig_trial    = shared_indices_h0[i];
        const int input_offset  = orig_trial * nbins;
        const int output_offset = i * nbins;
        for (int j = 0; j < nbins; ++j) {
            folds_h0_pruned[output_offset + j] = folds_h0_sim[input_offset + j];
        }
    }

    // Copy H1 surviving trials
    for (int i = tid; i < shared_count_h1; i += block_size) {
        const int orig_trial    = shared_indices_h1[i];
        const int input_offset  = orig_trial * nbins;
        const int output_offset = i * nbins;
        for (int j = 0; j < nbins; ++j) {
            folds_h1_pruned[output_offset + j] = folds_h1_sim[input_offset + j];
        }
    }

    // Store final counts (single thread)
    if (tid == 0) {
        *output_ntrials_h0 = shared_count_h0;
        *output_ntrials_h1 = shared_count_h1;
    }
}

__global__ void process_transitions_unified_kernel(
    const TransitionWorkItem* __restrict__ work_items,
    int num_items,
    const float* __restrict__ profile,
    int nbins,
    const SizeType* __restrict__ box_score_widths,
    int nwidths,
    float bias_snr,
    float var_add,
    const float* __restrict__ probs,
    int nprobs,
    TransitionResult* __restrict__ results,
    uint64_t seed,
    uint64_t offset) {

    const auto item_idx = static_cast<int>(blockIdx.x);
    if (item_idx >= num_items) {
        return;
    }

    const auto& work_item = work_items[item_idx];
    const auto tid        = static_cast<int>(threadIdx.x);

    // Shared memory for output counts
    __shared__ int shared_ntrials_h0_out;
    __shared__ int shared_ntrials_h1_out;

    // Phase 1: Simulation (threads collaborate)
    simulate_transition_phase(work_item, profile, nbins, bias_snr, var_add,
                              seed, offset);
    __syncthreads();

    // Phase 2: Fused Score and Prune (threads collaborate)
    score_and_prune_fused(work_item, box_score_widths, nwidths, nbins,
                          work_item.threshold, &shared_ntrials_h0_out,
                          &shared_ntrials_h1_out);
    __syncthreads();

    // Phase 3: Compute final state and result (single thread per block)
    if (tid == 0) {
        const auto ntrials_h0_out = shared_ntrials_h0_out;
        const auto ntrials_h1_out = shared_ntrials_h1_out;
        // Calculate success rates
        const auto success_h0 =
            static_cast<float>(ntrials_h0_out) /
            static_cast<float>(work_item.folds_sim.folds_h0.ntrials);
        const auto success_h1 =
            static_cast<float>(ntrials_h1_out) /
            static_cast<float>(work_item.folds_sim.folds_h1.ntrials);

        // Generate next state
        const auto state_next = work_item.input_state.gen_next(
            work_item.threshold, success_h0, success_h1, work_item.nbranches);

        // Find probability bin
        const auto iprob =
            find_bin_index_device(probs, nprobs, state_next.success_h1_cumul);

        // Store result
        TransitionResult& result = results[item_idx];
        result.threshold_idx     = work_item.threshold_idx;
        result.prob_idx          = iprob;
        result.computed_state    = state_next;
        result.folds_out         = work_item.folds_pruned;
        result.invalid           = (iprob < 0 || iprob >= nprobs);
    }
}

__global__ void
update_states_kernel(const TransitionResult* results,
                     int num_results,
                     int nprobs,
                     int stage_offset_cur,
                     StateD* states_out_ptr,
                     cuda::std::optional<FoldsTypeDevice>* folds_out_ptr) {

    const auto idx = static_cast<int>((blockIdx.x * blockDim.x) + threadIdx.x);
    if (idx >= num_results) {
        return;
    }
    const auto& result = results[idx];
    if (result.invalid) {
        return;
    }
    const int fold_idx  = (result.threshold_idx * nprobs) + result.prob_idx;
    const int state_idx = stage_offset_cur + fold_idx;
    // Use atomic compare-and-swap to handle race conditions
    // Convert float to int for atomic operations (assumes IEEE 754)
    const int new_complexity_bits =
        __float_as_int(result.computed_state.complexity_cumul);
    int* state_complexity_ptr =
        reinterpret_cast<int*>(&states_out_ptr[state_idx].complexity_cumul);

    // Atomic compare-and-swap loop
    int old_complexity_bits = atomicOr(state_complexity_ptr, 0);

    bool should_update = false;
    if (states_out_ptr[state_idx].is_empty) {
        // Try to claim this empty slot
        int expected_empty = __float_as_int(0.0F);
        if (atomicCAS(state_complexity_ptr, expected_empty,
                      new_complexity_bits) == expected_empty) {
            should_update = true;
        }
    } else {
        // Compare complexities and try to update if we're better
        float old_complexity = __int_as_float(old_complexity_bits);
        if (result.computed_state.complexity_cumul < old_complexity) {
            if (atomicCAS(state_complexity_ptr, old_complexity_bits,
                          new_complexity_bits) == old_complexity_bits) {
                should_update = true;
            }
        }
    }
    if (should_update) {
        states_out_ptr[state_idx] = result.computed_state;
        folds_out_ptr[fold_idx]   = result.folds_out;
    }
}

struct IndexPair {
    SizeType ithres;
    SizeType jthresh;
};

struct PairWithCount {
    IndexPair pair;
    SizeType count; // Number of valid kprob iterations
};

struct CountValidWorkItems {
    const StateD* states_ptr;
    const cuda::std::optional<FoldsTypeDevice>* folds_ptr;
    SizeType stage_offset_prev;
    SizeType nprobs;

    CountValidWorkItems(
        const thrust::device_vector<StateD>& states_d,
        const thrust::device_vector<cuda::std::optional<FoldsTypeDevice>>&
            folds_current_d,
        SizeType stage_offset_prev,
        SizeType nprobs)
        : states_ptr(thrust::raw_pointer_cast(states_d.data())),
          folds_ptr(thrust::raw_pointer_cast(folds_current_d.data())),
          stage_offset_prev(stage_offset_prev),
          nprobs(nprobs) {}

    __device__ PairWithCount operator()(const IndexPair& pair) const {
        SizeType count = 0;
        for (SizeType kprob = 0; kprob < nprobs; ++kprob) {
            const auto prev_fold_idx = (pair.jthresh * nprobs) + kprob;
            const auto& prev_state =
                states_ptr[stage_offset_prev + prev_fold_idx];
            if (prev_state.is_empty) {
                continue;
            }
            const auto& prev_fold_state = folds_ptr[prev_fold_idx];
            if (!prev_fold_state.has_value() || prev_fold_state->is_empty()) {
                continue;
            }
            count++;
        }
        return {.pair = pair, .count = count};
    }
};

struct TransitionFunctor {
    const StateD* states_ptr;
    const cuda::std::optional<FoldsTypeDevice>* folds_ptr;
    const float* thresholds_ptr;
    const float* branching_pattern_ptr;
    SizeType nprobs;
    SizeType ntrials;
    SizeType stage_offset_prev;
    SizeType istage;
    float var_add;
    TransitionWorkItem* work_items_ptr;
    const SizeType* offset_ptr;
    DevicePoolAllocator* allocator;

    TransitionFunctor(
        const thrust::device_vector<StateD>& states_d,
        const thrust::device_vector<cuda::std::optional<FoldsTypeDevice>>&
            folds_current_d,
        const thrust::device_vector<float>& thresholds,
        const thrust::device_vector<float>& branching_pattern,
        SizeType nprobs,
        SizeType ntrials,
        SizeType offset_prev,
        SizeType stage,
        float var,
        thrust::device_vector<TransitionWorkItem>& items_d,
        const thrust::device_vector<SizeType>& offsets,
        DevicePoolAllocator* allocator)
        : states_ptr(thrust::raw_pointer_cast(states_d.data())),
          folds_ptr(thrust::raw_pointer_cast(folds_current_d.data())),
          thresholds_ptr(thrust::raw_pointer_cast(thresholds.data())),
          branching_pattern_ptr(
              thrust::raw_pointer_cast(branching_pattern.data())),
          nprobs(nprobs),
          ntrials(ntrials),
          stage_offset_prev(offset_prev),
          istage(stage),
          var_add(var),
          work_items_ptr(thrust::raw_pointer_cast(items_d.data())),
          offset_ptr(thrust::raw_pointer_cast(offsets.data())),
          allocator(allocator) {}

    __device__ void operator()(const PairWithCount& pair_with_count) const {
        const auto& pair = pair_with_count.pair;
        SizeType ithres  = pair.ithres;
        SizeType jthresh = pair.jthresh;
        // Current offset
        SizeType base_offset =
            offset_ptr[&pair_with_count - &pair_with_count[0]];
        SizeType slot = base_offset;
        for (SizeType kprob = 0; kprob < nprobs; ++kprob) {
            const auto prev_fold_idx = (jthresh * nprobs) + kprob;
            const auto& prev_state =
                states_ptr[stage_offset_prev + prev_fold_idx];
            if (prev_state.is_empty) {
                continue;
            }
            const auto& prev_fold_state = folds_ptr[prev_fold_idx];
            if (!prev_fold_state.has_value() || prev_fold_state->is_empty()) {
                continue;
            }
            // Pre-allocate output buffers
            const auto ntrials_in_h0 = prev_fold_state->folds_h0.ntrials;
            const auto ntrials_in_h1 = prev_fold_state->folds_h1.ntrials;
            const auto repeat_factor_h0 =
                static_cast<SizeType>(ceilf(static_cast<float>(ntrials) /
                                            static_cast<float>(ntrials_in_h0)));
            const auto ntrials_out_h0 = repeat_factor_h0 * ntrials_in_h0;
            const auto repeat_factor_h1 =
                static_cast<SizeType>(ceilf(static_cast<float>(ntrials) /
                                            static_cast<float>(ntrials_in_h1)));
            const auto ntrials_out_h1 = repeat_factor_h1 * ntrials_in_h1;

            auto folds_h0_sim = allocator->allocate(
                ntrials_out_h0, prev_fold_state->folds_h0.variance + var_add);
            auto folds_h1_sim = allocator->allocate(
                ntrials_out_h1, prev_fold_state->folds_h1.variance + var_add);
            auto folds_h0_prn =
                allocator->allocate(ntrials_out_h0, folds_h0_sim.variance);
            auto folds_h1_prn =
                allocator->allocate(ntrials_out_h1, folds_h1_sim.variance);

            // Populate the TransitionWorkItem
            TransitionWorkItem item;
            item.threshold_idx   = static_cast<int>(ithres);
            item.prob_idx        = static_cast<int>(kprob);
            item.input_fold_idx  = static_cast<int>(prev_fold_idx);
            item.input_state     = prev_state;
            item.threshold       = thresholds_ptr[ithres];
            item.nbranches       = branching_pattern_ptr[istage];
            item.folds_in        = FoldsTypeDevice(prev_fold_state->folds_h0,
                                                   prev_fold_state->folds_h1);
            item.folds_sim       = FoldsTypeDevice(folds_h0_sim, folds_h1_sim);
            item.folds_pruned    = FoldsTypeDevice(folds_h0_prn, folds_h1_prn);
            work_items_ptr[slot] = item;
            slot++;
        }
    }
};

// Create a compound type for State
HighFive::CompoundType create_compound_state() {
    return {{"success_h0", HighFive::create_datatype<float>()},
            {"success_h1", HighFive::create_datatype<float>()},
            {"complexity", HighFive::create_datatype<float>()},
            {"complexity_cumul", HighFive::create_datatype<float>()},
            {"success_h1_cumul", HighFive::create_datatype<float>()},
            {"nbranches", HighFive::create_datatype<float>()},
            {"threshold", HighFive::create_datatype<float>()},
            {"cost", HighFive::create_datatype<float>()},
            {"threshold_prev", HighFive::create_datatype<float>()},
            {"success_h1_cumul_prev", HighFive::create_datatype<float>()},
            {"is_empty", HighFive::create_datatype<bool>()}};
}

} // namespace

__host__ __device__ StateD StateD::gen_next(float threshold,
                                            float success_h0,
                                            float success_h1,
                                            float nbranches) const noexcept {
    const auto nleaves_next          = this->complexity * nbranches;
    const auto nleaves_surv          = nleaves_next * success_h0;
    const auto complexity_cumul_next = this->complexity_cumul + nleaves_next;
    const auto success_h1_cumul_next = this->success_h1_cumul * success_h1;
    const auto cost_next = complexity_cumul_next / success_h1_cumul_next;

    // Create a new state struct
    StateD state_next;
    state_next.success_h0       = success_h0;
    state_next.success_h1       = success_h1;
    state_next.complexity       = nleaves_surv;
    state_next.complexity_cumul = complexity_cumul_next;
    state_next.success_h1_cumul = success_h1_cumul_next;
    state_next.nbranches        = nbranches;
    state_next.threshold        = threshold;
    state_next.cost             = cost_next;
    state_next.is_empty         = false;
    // For backtracking
    state_next.threshold_prev        = this->threshold;
    state_next.success_h1_cumul_prev = this->success_h1_cumul;
    return state_next;
}

// CUDA-specific implementation
class DynamicThresholdSchemeCUDA::Impl {
public:
    Impl(std::span<const float> branching_pattern,
         float ref_ducy,
         SizeType nbins,
         SizeType ntrials,
         SizeType nprobs,
         float prob_min,
         float snr_final,
         SizeType nthresholds,
         float ducy_max,
         float wtsp,
         float beam_width,
         SizeType trials_start,
         int device_id)
        : m_branching_pattern(branching_pattern.begin(),
                              branching_pattern.end()),
          m_ref_ducy(ref_ducy),
          m_ntrials(ntrials),
          m_ducy_max(ducy_max),
          m_wtsp(wtsp),
          m_beam_width(beam_width),
          m_trials_start(trials_start),
          m_device_id(device_id) {

        cuda_utils::set_device(m_device_id);
        if (m_branching_pattern.empty()) {
            throw std::invalid_argument("Branching pattern is empty");
        }
        // Host-side computations
        m_profile = simulation::generate_folded_profile(nbins, ref_ducy);
        m_thresholds =
            detection::compute_thresholds(0.1F, snr_final, nthresholds);
        m_probs       = detection::compute_probs(nprobs, prob_min);
        m_nprobs      = m_probs.size();
        m_nbins       = m_profile.size();
        m_nstages     = m_branching_pattern.size();
        m_nthresholds = m_thresholds.size();
        m_box_score_widths =
            detection::generate_box_width_trials(m_nbins, m_ducy_max, m_wtsp);
        m_bias_snr   = snr_final / static_cast<float>(std::sqrt(m_nstages + 1));
        m_guess_path = detection::guess_scheme(
            m_nstages, snr_final, m_branching_pattern, m_trials_start);

        // Copy data to device
        m_branching_pattern_d = m_branching_pattern;
        m_profile_d           = m_profile;
        m_thresholds_d        = m_thresholds;
        m_probs_d             = m_probs;
        m_box_score_widths_d  = m_box_score_widths;

        m_rng = std::make_unique<math::CuRandRNG>();
        // Initialize memory management
        const auto slots_per_pool = compute_max_allocations_needed();
        m_device_manager          = std::make_unique<DualPoolFoldManagerDevice>(
            m_nbins, m_ntrials, slots_per_pool);
        spdlog::info("Pre-allocated 2 CUDA pools of {} slots each",
                     slots_per_pool);

        // Initialize state management
        m_folds_current_d.resize(m_nthresholds * m_nprobs);
        m_folds_next_d.resize(m_nthresholds * m_nprobs);
        m_states_d.resize(m_nstages * m_nthresholds * m_nprobs, StateD{});
        m_states.resize(m_nstages * m_nthresholds * m_nprobs, State{});
        init_states();
    }
    ~Impl()                          = default;
    Impl(const Impl&)                = delete;
    Impl& operator=(const Impl&)     = delete;
    Impl(Impl&&) noexcept            = default;
    Impl& operator=(Impl&&) noexcept = default;

    // Methods
    void run(SizeType thres_neigh = 10) {
        spdlog::info("Running dynamic threshold scheme on CUDA");
        utils::ProgressGuard progress_guard(true);
        auto bar = utils::make_standard_bar("Computing scheme...");

        for (SizeType istage = 1; istage < m_nstages; ++istage) {
            // Get device allocator for this stage
            auto allocator = m_device_manager->get_device_allocator();
            run_segment(istage, thres_neigh, allocator);
            m_device_manager->swap_pools();
            std::swap(m_folds_current_d, m_folds_next_d);
            // Deallocate using thrust
            thrust::for_each(thrust::device, m_folds_next_d.begin(),
                             m_folds_next_d.end(),
                             DeallocateFunctor{allocator});
            const auto progress = static_cast<float>(istage) /
                                  static_cast<float>(m_nstages - 1) * 100.0F;
            bar.set_progress(static_cast<SizeType>(progress));
        }
        // Copy final states back to host
        thrust::copy(thrust::device, m_states_d.begin(), m_states_d.end(),
                     m_states.begin());
    }

    std::string save(const std::string& outdir = "./") const {
        const std::filesystem::path filebase = std::format(
            "dynscheme_nstages_{:03d}_nthresh_{:03d}_nprobs_{:03d}_"
            "ntrials_{:04d}_snr_{:04.1f}_ducy_{:04.2f}_beam_{:03.1f}.h5",
            m_nstages, m_nthresholds, m_nprobs, m_ntrials, m_thresholds.back(),
            m_ref_ducy, m_beam_width);
        const std::filesystem::path filepath =
            std::filesystem::path(outdir) / filebase;
        HighFive::File file(filepath, HighFive::File::Overwrite);
        // Save simple attributes
        file.createAttribute("ntrials", m_ntrials);
        file.createAttribute("snr_final", m_thresholds.back());
        file.createAttribute("ref_ducy", m_ref_ducy);
        file.createAttribute("ducy_max", m_ducy_max);
        file.createAttribute("wtsp", m_wtsp);
        file.createAttribute("beam_width", m_beam_width);

        // Create dataset creation property list and enable compression
        HighFive::DataSetCreateProps props;
        props.add(HighFive::Chunking(std::vector<hsize_t>{1024}));
        props.add(HighFive::Deflate(9));

        // Save arrays
        file.createDataSet("branching_pattern", m_branching_pattern);
        file.createDataSet("profile", m_profile);
        file.createDataSet("thresholds", m_thresholds);
        file.createDataSet("probs", m_probs);
        file.createDataSet("guess_path", m_guess_path);
        // Define the 3D dataspace for states
        std::vector<SizeType> dims = {m_nstages, m_nthresholds, m_nprobs};
        HighFive::DataSetCreateProps props_states;
        std::vector<hsize_t> chunk_dims(dims.begin(), dims.end());
        props_states.add(HighFive::Chunking(chunk_dims));
        auto dataset =
            file.createDataSet("states", HighFive::DataSpace(dims),
                               create_compound_state(), props_states);
        dataset.write_raw(m_states.data());
        spdlog::info("Saved dynamic threshold scheme to {}", filepath.string());
        return filepath.string();
    }

private:
    // Host-side parameters and metadata
    std::vector<float> m_branching_pattern;
    float m_ref_ducy;
    SizeType m_ntrials;
    float m_ducy_max;
    float m_wtsp;
    float m_beam_width;
    SizeType m_trials_start;
    int m_device_id;

    std::vector<float> m_profile;
    std::vector<float> m_thresholds;
    std::vector<float> m_probs;
    SizeType m_nprobs;
    SizeType m_nbins;
    SizeType m_nstages;
    SizeType m_nthresholds;
    std::vector<SizeType> m_box_score_widths;
    float m_bias_snr;
    std::vector<float> m_guess_path;
    std::vector<State> m_states;

    // Memory and RNG management
    std::unique_ptr<DualPoolFoldManagerDevice> m_device_manager;
    std::unique_ptr<math::CuRandRNG> m_rng;

    // Device-side data
    thrust::device_vector<float> m_branching_pattern_d;
    thrust::device_vector<float> m_profile_d;
    thrust::device_vector<float> m_thresholds_d;
    thrust::device_vector<float> m_probs_d;
    thrust::device_vector<SizeType> m_box_score_widths_d;
    thrust::device_vector<StateD> m_states_d;
    thrust::device_vector<cuda::std::optional<FoldsTypeDevice>>
        m_folds_current_d;
    thrust::device_vector<cuda::std::optional<FoldsTypeDevice>> m_folds_next_d;

    SizeType compute_max_allocations_needed() {
        SizeType max_active_per_stage = 0;
        for (SizeType istage = 0; istage < m_nstages; ++istage) {
            auto active_thresholds = get_current_thresholds_idx(istage);
            max_active_per_stage =
                std::max(max_active_per_stage, active_thresholds.size());
        }
        // h0 + h1 per cell
        const auto max_persistent = max_active_per_stage * m_nprobs * 2;
        // 2 simulated and 2 pruned folds per transition
        const auto max_temporary  = 8; // Conservative for CUDA
        const auto slots_per_pool = max_persistent + max_temporary;
        spdlog::info(
            "CUDA allocation analysis: {} active thresholds max, {} prob "
            "bins",
            max_active_per_stage, m_nprobs);
        spdlog::info("Need {} persistent + {} temporary = {} slots per pool",
                     max_persistent, max_temporary, slots_per_pool);
        return slots_per_pool;
    }

    void init_states() {
        const float var_init = 1.0F;
        auto allocator       = m_device_manager->get_device_allocator();

        // Create initial batch for all thresholds in the initial beam
        const auto thresholds_idx = get_current_thresholds_idx(0);
        TransitionBatch initial_batch;
        initial_batch.reserve(thresholds_idx.size());

        // Allocate initial zero-filled fold vectors
        auto folds_h0_init = allocator.allocate(m_ntrials, 0.0F);
        auto folds_h1_init = allocator.allocate(m_ntrials, 0.0F);

        // Simulate the initial folds
        auto folds_h0_sim =
            simulate_folds(*folds_h0_init, m_profile_d_span, *m_rng, *m_manager,
                           0.0F, var_init, m_ntrials);
        auto folds_h1_sim =
            simulate_folds(*folds_h1_init, m_profile_d_span, *m_rng, *m_manager,
                           m_bias_snr, var_init, m_ntrials);

        // Create work items for each threshold in initial beam
        for (SizeType ithres : thresholds_idx) {
            // Allocate simulation and pruned output buffers
            auto folds_h0_sim    = allocator.allocate(m_ntrials, var_init);
            auto folds_h1_sim    = allocator.allocate(m_ntrials, var_init);
            auto folds_h0_pruned = allocator.allocate(m_ntrials, var_init);
            auto folds_h1_pruned = allocator.allocate(m_ntrials, var_init);

            TransitionWorkItem item;
            item.threshold_idx  = static_cast<int>(ithres);
            item.prob_idx       = -1; // Will be determined after processing
            item.input_fold_idx = -1; // Not applicable for initial state
            item.input_state    = StateD{}; // Empty initial state
            item.threshold      = m_thresholds[ithres];
            item.nbranches      = m_branching_pattern[0];
            item.folds_in       = FoldsTypeDevice(folds_h0_init, folds_h1_init);
            item.folds_sim      = FoldsTypeDevice(folds_h0_sim, folds_h1_sim);
            item.folds_pruned =
                FoldsTypeDevice(folds_h0_pruned, folds_h1_pruned);

            initial_batch.work_items_d.push_back(item);
        }

        // Process initial batch
        process_initial_batch(initial_batch, var_init);

        // Deallocate temporary initial buffers
        allocator.deallocate(folds_h0_init);
        allocator.deallocate(folds_h1_init);
    }

    std::vector<SizeType> get_current_thresholds_idx(SizeType istage) const {
        const auto guess       = m_guess_path[istage];
        const auto half_extent = m_beam_width;
        const auto lower_bound = std::max(0.0F, guess - half_extent);
        const auto upper_bound =
            std::min(m_thresholds.back(), guess + half_extent);

        std::vector<SizeType> result;
        for (SizeType i = 0; i < m_thresholds.size(); ++i) {
            if (m_thresholds[i] >= lower_bound &&
                m_thresholds[i] <= upper_bound) {
                result.push_back(i);
            }
        }
        return result;
    }

    void run_segment(SizeType istage,
                     SizeType thres_neigh,
                     DevicePoolAllocator allocator) {
        const float var_add          = 1.0F;
        const auto beam_idx_cur      = get_current_thresholds_idx(istage);
        const auto beam_idx_prev     = get_current_thresholds_idx(istage - 1);
        const auto stage_offset_prev = (istage - 1) * m_nthresholds * m_nprobs;
        const auto stage_offset_cur  = istage * m_nthresholds * m_nprobs;

        // Step 1: Collect all transitions into batch
        TransitionBatch batch;
        const SizeType max_transitions =
            beam_idx_cur.size() * thres_neigh * m_nprobs;
        batch.reserve(max_transitions);

        collect_transitions(batch, beam_idx_cur, beam_idx_prev,
                            stage_offset_prev, istage, thres_neigh, var_add,
                            allocator);

        if (batch.work_items_d.empty()) {
            return;
        }
        process_transition_batch(batch, var_add, stage_offset_cur);
    }

    void process_initial_batch(TransitionBatch& batch, float var_init) {
        const auto num_items = static_cast<int>(batch.work_items_d.size());
        if (num_items == 0) {
            return;
        }

        const uint64_t seed   = std::random_device{}();
        const uint64_t offset = 0;

        // Launch unified kernel for initial processing
        const dim3 block_dim(256);
        const dim3 grid_dim(num_items);
        // simulate phase and score_and_prune phase
        const SizeType shared_mem_size = std::max(
            {static_cast<SizeType>(m_nbins * sizeof(float)),
             static_cast<SizeType>(2 * (1 + block_dim.x) * sizeof(int))});
        cuda_utils::check_kernel_launch_params(grid_dim, block_dim,
                                               shared_mem_size);

        process_transitions_unified_kernel<<<grid_dim, block_dim,
                                             shared_mem_size>>>(
            thrust::raw_pointer_cast(batch.work_items_d.data()), num_items,
            thrust::raw_pointer_cast(m_profile_d.data()),
            static_cast<int>(m_nbins),
            thrust::raw_pointer_cast(m_box_score_widths_d.data()),
            static_cast<int>(m_box_score_widths_d.size()), m_bias_snr, var_init,
            thrust::raw_pointer_cast(m_probs_d.data()),
            static_cast<int>(m_nprobs),
            thrust::raw_pointer_cast(batch.results_d.data()), seed, offset);

        cuda_utils::check_last_cuda_error("process_initial_batch");

        // Process results and populate initial states
        std::vector<TransitionResult> results_h(num_items);
        thrust::copy(batch.results_d.begin(), batch.results_d.end(),
                     results_h.begin());

        for (const auto& result : results_h) {
            if (result.invalid) {
                continue;
            }

            const auto fold_idx =
                (result.threshold_idx * m_nprobs) + result.prob_idx;
            m_states_d[fold_idx]        = result.computed_state;
            m_folds_current_d[fold_idx] = result.folds_out;
        }
    }

    void collect_transitions(TransitionBatch& batch,
                             const std::vector<SizeType>& beam_idx_cur,
                             const std::vector<SizeType>& beam_idx_prev,
                             SizeType stage_offset_prev,
                             SizeType istage,
                             SizeType thres_neigh,
                             float var_add,
                             DevicePoolAllocator& allocator) {
        std::vector<IndexPair> h_pairs;
        h_pairs.reserve(beam_idx_cur.size() * thres_neigh);
        for (SizeType i = 0; i < beam_idx_cur.size(); ++i) {
            const auto ithres = beam_idx_cur[i];
            // Find nearest neighbors in the previous beam
            const auto neighbour_beam_indices =
                utils::find_neighbouring_indices(beam_idx_prev, ithres,
                                                 thres_neigh);

            for (SizeType jthresh : neighbour_beam_indices) {
                h_pairs.push_back({ithres, jthresh});
            }
        }
        thrust::device_vector<IndexPair> d_pairs = h_pairs;
        thrust::device_vector<PairWithCount> d_pairs_with_count(d_pairs.size());
        thrust::transform(thrust::device, d_pairs.begin(), d_pairs.end(),
                          d_pairs_with_count.begin(),
                          CountValidWorkItems(m_states_d, m_folds_current_d,
                                              stage_offset_prev, m_nprobs));

        thrust::device_vector<SizeType> d_offsets(d_pairs.size());
        thrust::transform(
            thrust::device, d_pairs_with_count.begin(),
            d_pairs_with_count.end(), d_offsets.begin(),
            [] __device__(const PairWithCount& pwc) { return pwc.count; });
        thrust::exclusive_scan(thrust::device, d_offsets.begin(),
                               d_offsets.end(), d_offsets.begin());

        SizeType total_items = 0;
        if (!d_offsets.empty()) {
            const auto count =
                thrust::raw_pointer_cast(
                    d_pairs_with_count.data())[d_pairs_with_count.size() - 1]
                    .count;
            total_items = d_offsets.back() + count;
        }
        batch.work_items_d.resize(total_items);

        thrust::for_each(
            thrust::device, d_pairs_with_count.begin(),
            d_pairs_with_count.end(),
            TransitionFunctor(m_states_d, m_folds_current_d, m_thresholds_d,
                              m_branching_pattern_d, m_nprobs, m_ntrials,
                              stage_offset_prev, istage, var_add,
                              batch.work_items_d, d_offsets, &allocator));

        hipDeviceSynchronize();
    }

    void process_transition_batch(TransitionBatch& batch,
                                  float var_add,
                                  SizeType stage_offset_cur) {
        const auto num_items = static_cast<int>(batch.work_items_d.size());
        if (num_items == 0) {
            return;
        }

        // Generate random seed and offset
        const uint64_t seed   = std::random_device{}();
        const uint64_t offset = 0;
        // Launch unified kernel: one block per transition
        const dim3 block_dim(256);
        const dim3 grid_dim(num_items);
        // simulate phase and score_and_prune phase
        const SizeType shared_mem_size = std::max(
            {static_cast<SizeType>(m_nbins * sizeof(float)),
             static_cast<SizeType>(2 * (1 + block_dim.x) * sizeof(int))});
        cuda_utils::check_kernel_launch_params(grid_dim, block_dim,
                                               shared_mem_size);

        process_transitions_unified_kernel<<<grid_dim, block_dim,
                                             shared_mem_size>>>(
            thrust::raw_pointer_cast(batch.work_items_d.data()), num_items,
            thrust::raw_pointer_cast(m_profile_d.data()),
            static_cast<int>(m_nbins),
            thrust::raw_pointer_cast(m_box_score_widths_d.data()),
            static_cast<int>(m_box_score_widths_d.size()), m_bias_snr, var_add,
            thrust::raw_pointer_cast(m_probs_d.data()),
            static_cast<int>(m_nprobs),
            thrust::raw_pointer_cast(batch.results_d.data()), seed, offset);
        cuda_utils::check_last_cuda_error("process_transitions_unified_kernel");

        // Phase 2: Launch state update kernel
        const dim3 update_block_dim(256);
        const dim3 update_grid_dim((num_items + update_block_dim.x - 1) /
                                   update_block_dim.x);
        cuda_utils::check_kernel_launch_params(update_grid_dim,
                                               update_block_dim);

        update_states_kernel<<<update_grid_dim, update_block_dim>>>(
            thrust::raw_pointer_cast(batch.results_d.data()), num_items,
            static_cast<int>(m_nprobs), static_cast<int>(stage_offset_cur),
            thrust::raw_pointer_cast(m_states_d.data()),
            thrust::raw_pointer_cast(m_folds_next_d.data()));
        cuda_utils::check_last_cuda_error("update_states_kernel");
    }
};

DynamicThresholdSchemeCUDA::DynamicThresholdSchemeCUDA(
    std::span<const float> branching_pattern,
    float ref_ducy,
    SizeType nbins,
    SizeType ntrials,
    SizeType nprobs,
    float prob_min,
    float snr_final,
    SizeType nthresholds,
    float ducy_max,
    float wtsp,
    float beam_width,
    SizeType trials_start,
    int device_id)
    : m_impl(std::make_unique<Impl>(branching_pattern,
                                    ref_ducy,
                                    nbins,
                                    ntrials,
                                    nprobs,
                                    prob_min,
                                    snr_final,
                                    nthresholds,
                                    ducy_max,
                                    wtsp,
                                    beam_width,
                                    trials_start,
                                    device_id)) {}
DynamicThresholdSchemeCUDA::~DynamicThresholdSchemeCUDA() = default;
DynamicThresholdSchemeCUDA::DynamicThresholdSchemeCUDA(
    DynamicThresholdSchemeCUDA&&) noexcept = default;
DynamicThresholdSchemeCUDA& DynamicThresholdSchemeCUDA::operator=(
    DynamicThresholdSchemeCUDA&&) noexcept = default;

void DynamicThresholdSchemeCUDA::run(SizeType thres_neigh) {
    m_impl->run(thres_neigh);
}
std::string DynamicThresholdSchemeCUDA::save(const std::string& outdir) const {
    return m_impl->save(outdir);
}

} // namespace loki::detection

HIGHFIVE_REGISTER_TYPE(loki::detection::State,
                       loki::detection::create_compound_state)