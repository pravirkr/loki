#include "hip/hip_runtime.h"
#include "loki/utils/fft.hpp"

#include <format>
#include <spdlog/spdlog.h>

#include <cuda/std/complex>
#include <cuda/std/span>
#include <hip/hip_runtime_api.h>
#include <hipfft/hipfft.h>
#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>
#include <thrust/transform.h>

#include "loki/cuda_utils.cuh"

namespace loki::utils {

void rfft_batch_cuda(cuda::std::span<float> real_input,
                     cuda::std::span<ComplexTypeCUDA> complex_output,
                     int batch_size,
                     int n_real,
                     hipStream_t stream) {
    int n_complex = (n_real / 2) + 1;

    // Input validation
    if (static_cast<int>(real_input.size()) != batch_size * n_real) {
        throw std::runtime_error(
            std::format("RFFT CUDA batch: real_input size mismatch. Expected "
                        "{}, got {}",
                        batch_size * n_real, real_input.size()));
    }
    if (static_cast<int>(complex_output.size()) != batch_size * n_complex) {
        throw std::runtime_error(std::format(
            "RFFT CUDA batch: complex_output size mismatch. Expected "
            "{}, got {}",
            batch_size * n_complex, complex_output.size()));
    }
    auto* real_ptr    = real_input.data();
    auto* complex_ptr = reinterpret_cast<hipfftComplex*>(complex_output.data());

    hipfftHandle plan;
    cuda_utils::check_cuda_call(
        hipfftPlan1d(&plan, n_real, HIPFFT_R2C, batch_size),
        "RFFT CUDA: hipfftPlan1d failed");
    if (stream != nullptr) {
        cuda_utils::check_cuda_call(hipfftSetStream(plan, stream),
                                    "RFFT CUDA: hipfftSetStream failed");
    }
    cuda_utils::check_cuda_call(hipfftExecR2C(plan, real_ptr, complex_ptr),
                                "RFFT CUDA: hipfftExecR2C failed");
    cuda_utils::check_cuda_call(hipfftDestroy(plan),
                                "RFFT CUDA: hipfftDestroy failed");
    spdlog::debug("RFFT CUDA batch completed: {} transforms of size {}",
                  batch_size, n_real);
}

void irfft_batch_cuda(cuda::std::span<ComplexTypeCUDA> complex_input,
                      cuda::std::span<float> real_output,
                      int batch_size,
                      int n_real,
                      hipStream_t stream) {
    const int n_complex = (n_real / 2) + 1;

    // Input validation
    if (static_cast<int>(complex_input.size()) != batch_size * n_complex) {
        throw std::runtime_error(std::format(
            "IRFFT CUDA batch: complex_input size mismatch. Expected "
            "{}, got {}",
            batch_size * n_complex, complex_input.size()));
    }
    if (static_cast<int>(real_output.size()) != batch_size * n_real) {
        throw std::runtime_error(
            std::format("IRFFT CUDA batch: real_output size mismatch. Expected "
                        "{}, got {}",
                        batch_size * n_real, real_output.size()));
    }

    auto* complex_ptr = reinterpret_cast<hipfftComplex*>(complex_input.data());
    auto* real_ptr    = real_output.data();

    // Create cuFFT plan
    hipfftHandle plan;
    cuda_utils::check_cuda_call(
        hipfftPlan1d(&plan, n_real, HIPFFT_C2R, batch_size),
        "IRFFT CUDA: hipfftPlan1d failed");
    if (stream != nullptr) {
        cuda_utils::check_cuda_call(hipfftSetStream(plan, stream),
                                    "IRFFT CUDA: hipfftSetStream failed");
    }
    cuda_utils::check_cuda_call(hipfftExecC2R(plan, complex_ptr, real_ptr),
                                "IRFFT CUDA: hipfftExecC2R failed");
    cuda_utils::check_cuda_call(hipfftDestroy(plan),
                                "IRFFT CUDA: hipfftDestroy failed");
    // Apply normalization (cuFFT C2R doesn't normalize automatically)
    const float norm         = 1.0F / static_cast<float>(n_real);
    const int total_elements = batch_size * n_real;

    thrust::transform(thrust::device, real_ptr, real_ptr + total_elements,
                      real_ptr,
                      [norm] __device__(float x) { return x * norm; });

    spdlog::debug("IRFFT CUDA batch completed: {} transforms of size {}",
                  batch_size, n_real);
}

} // namespace loki::utils