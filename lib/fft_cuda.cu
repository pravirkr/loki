#include "hip/hip_runtime.h"
#include "loki/utils/fft.hpp"

#include <format>
#include <spdlog/spdlog.h>

#include <cuda/std/complex>
#include <cuda/std/span>
#include <hip/hip_runtime_api.h>
#include <hipfft/hipfft.h>
#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>
#include <thrust/transform.h>

#include "loki/cuda_utils.cuh"
#include "loki/exceptions.hpp"

namespace loki::utils {

void rfft_batch_cuda(cuda::std::span<float> real_input,
                     cuda::std::span<ComplexTypeCUDA> complex_output,
                     int batch_size,
                     int n_real,
                     hipStream_t stream) {
    int n_complex = (n_real / 2) + 1;

    // Input validation
    error_check::check_equal(
        real_input.size(), batch_size * n_real,
        "RFFT CUDA batch: real_input size does not match batch size");
    error_check::check_equal(
        complex_output.size(), batch_size * n_complex,
        "RFFT CUDA batch: complex_output size does not match batch size");

    auto* real_ptr    = real_input.data();
    auto* complex_ptr = reinterpret_cast<hipfftComplex*>(complex_output.data());

    hipfftHandle plan;
    cuda_utils::check_cuda_call(
        hipfftPlan1d(&plan, n_real, HIPFFT_R2C, batch_size),
        "RFFT CUDA: hipfftPlan1d failed");
    if (stream != nullptr) {
        cuda_utils::check_cuda_call(hipfftSetStream(plan, stream),
                                    "RFFT CUDA: hipfftSetStream failed");
    }
    cuda_utils::check_cuda_call(hipfftExecR2C(plan, real_ptr, complex_ptr),
                                "RFFT CUDA: hipfftExecR2C failed");
    cuda_utils::check_cuda_call(hipfftDestroy(plan),
                                "RFFT CUDA: hipfftDestroy failed");
    spdlog::debug("RFFT CUDA batch completed: {} transforms of size {}",
                  batch_size, n_real);
}

void rfft_batch_inplace_cuda(cuda::std::span<ComplexTypeCUDA> inout_buffer,
                             int batch_size,
                             int n_real,
                             hipStream_t stream) {
    // Input validation
    if (batch_size <= 0 || n_real <= 0) {
        throw std::invalid_argument(
            std::format("RFFT CUDA in-place: batch_size ({}) and n_real ({}) "
                        "must be positive",
                        batch_size, n_real));
    }
    if (n_real % 2 != 0) {
        throw std::invalid_argument(std::format(
            "RFFT CUDA in-place: n_real ({}) must be even", n_real));
    }

    const int n_complex = (n_real / 2) + 1;

    // For in-place R2C, cuFFT needs space for n_real + 2 floats per batch
    const int required_floats  = batch_size * (n_real + 2);
    const int available_floats = static_cast<int>(inout_buffer.size()) * 2;
    if (available_floats < required_floats) {
        throw std::invalid_argument(
            std::format("RFFT CUDA in-place: buffer too small. Need {} floats "
                        "({} complex), "
                        "have {} floats ({} complex)",
                        required_floats, (required_floats + 1) / 2,
                        available_floats, inout_buffer.size()));
    }

    // Reinterpret buffer for cuFFT
    auto* real_ptr    = reinterpret_cast<float*>(inout_buffer.data());
    auto* complex_ptr = reinterpret_cast<hipfftComplex*>(inout_buffer.data());

    // Create cuFFT plan for in-place R2C
    hipfftHandle plan;
    cuda_utils::check_cuda_call(
        hipfftPlan1d(&plan, n_real, HIPFFT_R2C, batch_size),
        "RFFT CUDA in-place: hipfftPlan1d failed");

    if (stream != nullptr) {
        cuda_utils::check_cuda_call(
            hipfftSetStream(plan, stream),
            "RFFT CUDA in-place: hipfftSetStream failed");
    }

    // Execute in-place R2C transform
    cuda_utils::check_cuda_call(hipfftExecR2C(plan, real_ptr, complex_ptr),
                                "RFFT CUDA in-place: hipfftExecR2C failed");

    cuda_utils::check_cuda_call(hipfftDestroy(plan),
                                "RFFT CUDA in-place: hipfftDestroy failed");

    spdlog::debug(
        "RFFT CUDA in-place batch completed: {} transforms of size {}",
        batch_size, n_real);
}

void irfft_batch_cuda(cuda::std::span<ComplexTypeCUDA> complex_input,
                      cuda::std::span<float> real_output,
                      int batch_size,
                      int n_real,
                      hipStream_t stream) {
    const int n_complex = (n_real / 2) + 1;

    // Input validation
    error_check::check_equal(
        real_output.size(), batch_size * n_real,
        "IRFFT CUDA batch: real_output size does not match batch size");
    error_check::check_equal(
        complex_input.size(), batch_size * n_complex,
        "IRFFT CUDA batch: complex_input size does not match batch size");

    auto* complex_ptr = reinterpret_cast<hipfftComplex*>(complex_input.data());
    auto* real_ptr    = real_output.data();

    // Create cuFFT plan
    hipfftHandle plan;
    cuda_utils::check_cuda_call(
        hipfftPlan1d(&plan, n_real, HIPFFT_C2R, batch_size),
        "IRFFT CUDA: hipfftPlan1d failed");
    if (stream != nullptr) {
        cuda_utils::check_cuda_call(hipfftSetStream(plan, stream),
                                    "IRFFT CUDA: hipfftSetStream failed");
    }
    cuda_utils::check_cuda_call(hipfftExecC2R(plan, complex_ptr, real_ptr),
                                "IRFFT CUDA: hipfftExecC2R failed");
    cuda_utils::check_cuda_call(hipfftDestroy(plan),
                                "IRFFT CUDA: hipfftDestroy failed");
    // Apply normalization (cuFFT C2R doesn't normalize automatically)
    const float norm         = 1.0F / static_cast<float>(n_real);
    const int total_elements = batch_size * n_real;

    // Use thrust for efficient GPU normalization
    if (stream != nullptr) {
        thrust::transform(thrust::cuda::par.on(stream), real_ptr,
                          real_ptr + total_elements, real_ptr,
                          [norm] __device__(float x) { return x * norm; });
    } else {
        thrust::transform(thrust::device, real_ptr, real_ptr + total_elements,
                          real_ptr,
                          [norm] __device__(float x) { return x * norm; });
    }

    spdlog::debug("IRFFT CUDA batch completed: {} transforms of size {}",
                  batch_size, n_real);
}

void irfft_batch_inplace_cuda(cuda::std::span<ComplexTypeCUDA> inout_buffer,
                              int batch_size,
                              int n_real,
                              hipStream_t stream) {
    // Input validation
    if (batch_size <= 0 || n_real <= 0) {
        throw std::invalid_argument(
            std::format("IRFFT CUDA in-place: batch_size ({}) and n_real ({}) "
                        "must be positive",
                        batch_size, n_real));
    }
    if (n_real % 2 != 0) {
        throw std::invalid_argument(std::format(
            "IRFFT CUDA in-place: n_real ({}) must be even", n_real));
    }

    const int n_complex = (n_real / 2) + 1;

    // Check minimum buffer size for complex input
    const int required_complex = batch_size * n_complex;
    if (static_cast<int>(inout_buffer.size()) < required_complex) {
        throw std::invalid_argument(
            std::format("IRFFT CUDA in-place: buffer too small. Need {} "
                        "complex elements, have {}",
                        required_complex, inout_buffer.size()));
    }

    // Reinterpret buffer for cuFFT
    auto* complex_ptr = reinterpret_cast<hipfftComplex*>(inout_buffer.data());
    auto* real_ptr    = reinterpret_cast<float*>(inout_buffer.data());

    // Create cuFFT plan for in-place C2R
    hipfftHandle plan;
    cuda_utils::check_cuda_call(
        hipfftPlan1d(&plan, n_real, HIPFFT_C2R, batch_size),
        "IRFFT CUDA in-place: hipfftPlan1d failed");

    if (stream != nullptr) {
        cuda_utils::check_cuda_call(
            hipfftSetStream(plan, stream),
            "IRFFT CUDA in-place: hipfftSetStream failed");
    }

    // Execute in-place C2R transform
    cuda_utils::check_cuda_call(hipfftExecC2R(plan, complex_ptr, real_ptr),
                                "IRFFT CUDA in-place: hipfftExecC2R failed");

    cuda_utils::check_cuda_call(hipfftDestroy(plan),
                                "IRFFT CUDA in-place: hipfftDestroy failed");

    // Apply normalization (cuFFT C2R doesn't normalize automatically)
    const float norm         = 1.0F / static_cast<float>(n_real);
    const int total_elements = batch_size * n_real;

    // Use thrust for efficient GPU normalization
    if (stream != nullptr) {
        thrust::transform(thrust::cuda::par.on(stream), real_ptr,
                          real_ptr + total_elements, real_ptr,
                          [norm] __device__(float x) { return x * norm; });
    } else {
        thrust::transform(thrust::device, real_ptr, real_ptr + total_elements,
                          real_ptr,
                          [norm] __device__(float x) { return x * norm; });
    }

    spdlog::debug(
        "IRFFT CUDA in-place batch completed: {} transforms of size {}",
        batch_size, n_real);
}

} // namespace loki::utils