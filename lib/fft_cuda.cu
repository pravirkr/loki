#include "hip/hip_runtime.h"
#include "loki/utils/fft.hpp"

#include <format>
#include <spdlog/spdlog.h>

#include <cuda/std/complex>
#include <cuda/std/span>
#include <hip/hip_runtime_api.h>
#include <hipfft/hipfft.h>
#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>
#include <thrust/transform.h>

#include "loki/cuda_utils.cuh"
#include "loki/exceptions.hpp"

namespace loki::utils {

void rfft_batch_cuda(cuda::std::span<float> real_input,
                     cuda::std::span<ComplexTypeCUDA> complex_output,
                     int batch_size,
                     int n_real,
                     hipStream_t stream) {
    int n_complex = (n_real / 2) + 1;

    // Input validation
    error_check::check_equal(
        real_input.size(), batch_size * n_real,
        "RFFT CUDA batch: real_input size does not match batch size");
    error_check::check_equal(
        complex_output.size(), batch_size * n_complex,
        "RFFT CUDA batch: complex_output size does not match batch size");

    auto* real_ptr    = real_input.data();
    auto* complex_ptr = reinterpret_cast<hipfftComplex*>(complex_output.data());

    hipfftHandle plan;
    cuda_utils::check_cuda_call(
        hipfftPlan1d(&plan, n_real, HIPFFT_R2C, batch_size),
        "RFFT CUDA: hipfftPlan1d failed");
    if (stream != nullptr) {
        cuda_utils::check_cuda_call(hipfftSetStream(plan, stream),
                                    "RFFT CUDA: hipfftSetStream failed");
    }
    cuda_utils::check_cuda_call(hipfftExecR2C(plan, real_ptr, complex_ptr),
                                "RFFT CUDA: hipfftExecR2C failed");
    cuda_utils::check_cuda_call(hipfftDestroy(plan),
                                "RFFT CUDA: hipfftDestroy failed");
    spdlog::debug("RFFT CUDA batch completed: {} transforms of size {}",
                  batch_size, n_real);
}

void irfft_batch_cuda(cuda::std::span<ComplexTypeCUDA> complex_input,
                      cuda::std::span<float> real_output,
                      int batch_size,
                      int n_real,
                      hipStream_t stream) {
    const int n_complex = (n_real / 2) + 1;

    // Input validation
    error_check::check_equal(
        real_output.size(), batch_size * n_real,
        "IRFFT CUDA batch: real_output size does not match batch size");
    error_check::check_equal(
        complex_input.size(), batch_size * n_complex,
        "IRFFT CUDA batch: complex_input size does not match batch size");

    auto* complex_ptr = reinterpret_cast<hipfftComplex*>(complex_input.data());
    auto* real_ptr    = real_output.data();

    // Create cuFFT plan
    hipfftHandle plan;
    cuda_utils::check_cuda_call(
        hipfftPlan1d(&plan, n_real, HIPFFT_C2R, batch_size),
        "IRFFT CUDA: hipfftPlan1d failed");
    if (stream != nullptr) {
        cuda_utils::check_cuda_call(hipfftSetStream(plan, stream),
                                    "IRFFT CUDA: hipfftSetStream failed");
    }
    cuda_utils::check_cuda_call(hipfftExecC2R(plan, complex_ptr, real_ptr),
                                "IRFFT CUDA: hipfftExecC2R failed");
    cuda_utils::check_cuda_call(hipfftDestroy(plan),
                                "IRFFT CUDA: hipfftDestroy failed");
    // Apply normalization (cuFFT C2R doesn't normalize automatically)
    const float norm         = 1.0F / static_cast<float>(n_real);
    const int total_elements = batch_size * n_real;

    // Use thrust for efficient GPU normalization
    if (stream != nullptr) {
        thrust::transform(thrust::cuda::par.on(stream), real_ptr,
                          real_ptr + total_elements, real_ptr,
                          [norm] __device__(float x) { return x * norm; });
    } else {
        thrust::transform(thrust::device, real_ptr, real_ptr + total_elements,
                          real_ptr,
                          [norm] __device__(float x) { return x * norm; });
    }

    spdlog::debug("IRFFT CUDA batch completed: {} transforms of size {}",
                  batch_size, n_real);
}

} // namespace loki::utils